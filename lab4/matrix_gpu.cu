#include "hip/hip_runtime.h"
// Matrix addition, CPU version
// gcc matrix_cpu.c -o matrix_cpu -std=c99

#include <stdio.h>


void printDeviceProperties(){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("  Device name: %s\n", prop.name);
	printf("  Memory Clock Rate (KHz): %d\n",
				 prop.memoryClockRate);
	printf("  Memory Bus Width (bits): %d\n",
				 prop.memoryBusWidth);
	printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
				 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
}


__global__
void add_matrix(float *a, float *b, float *c, int N) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}


int main() {
	printDeviceProperties();
	const int N = 16;

	float a[N*N];
	float b[N*N];
	float c[N*N];
	float* ad;
	float* bd;
	float* cd;
	const int size = N * N * sizeof(float);
	hipMalloc((void**)&ad, size);
	hipMalloc((void**)&bd, size);
	hipMalloc((void**)&cd, size);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)	{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}
	}
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);

	dim3 dimBlock(N, N);
	dim3 dimGrid(8, 8);
	add_matrix<<<dimBlock, dimGrid>>>(ad, bd, cd, N);
	hipDeviceSynchronize();
	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}
}
