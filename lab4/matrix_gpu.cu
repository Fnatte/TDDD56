#include "hip/hip_runtime.h"
// Matrix addition, CPU version
// gcc matrix_cpu.c -o matrix_cpu -std=c99

#include <stdio.h>


void printDeviceProperties(){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("  Device name: %s\n", prop.name);
	printf("  Memory Clock Rate (KHz): %d\n",
				 prop.memoryClockRate);
	printf("  Memory Bus Width (bits): %d\n",
				 prop.memoryBusWidth);
	printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
				 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
}


__global__
void add_matrix(float *a, float *b, float *c, int N) {
	int indexX = blockIdx.x * blockDim.x + threadIdx.x;
	int indexY = blockIdx.y * blockDim.y + threadIdx.y;
	int index = indexY * N + indexX;
	c[index] = a[index] + b[index];
}


int main() {
	printDeviceProperties();
	const int N = 1024;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* a = new float[N*N];
	float* b = new float[N*N];
	float* c = new float[N*N];
	float* ad;
	float* bd;
	float* cd;
	const int size = N * N * sizeof(float);
	hipMalloc((void**)&ad, size);
	hipMalloc((void**)&bd, size);
	hipMalloc((void**)&cd, size);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)	{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}
	}
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);

	int gridDim = 16;
	dim3 dimBlock(N/gridDim, N/gridDim);
	dim3 dimGrid(gridDim, gridDim);
	hipEventRecord(start);
	add_matrix<<<dimBlock, dimGrid>>>(ad, bd, cd, N);
	hipEventRecord(stop);
	hipDeviceSynchronize();
	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);


	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}

	printf("Execution took %f milliseconds.\n", milliseconds);
}
